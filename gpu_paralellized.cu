#include "hip/hip_runtime.h"
#include<stdio.h>
#include "mysql_drivers/sqlite3.h"
#define SIZE 100
__global__ void create_db(const char* dir){
	sqlite3*  DB;
	int exit = 0;
	exit = sqlite3_open(dir,&DB);
	sqlite3_close(DB);
}
__global__ void create_table(const char*dir){
	sqlite3* DB;
	int exit = 0;
	exit = sqlite3_open(dir,&DB);

	char* query = " CREATE TABLE IF NOT EXISTS contacts ( \
		contact_id INTEGER PRIMARY KEY, \
		first_name TEXT NOT NULL, \
		last_name TEXT NOT NULL, \
		email TEXT NOT NULL UNIQUE, \
		phone TEXT NOT NULL UNIQUE );";

	char* message;
	
	exit = sqlite3_exec(DB,query,NULL,0,&message);
	if(message != NULL){
		printf("%s",message);
	}
	sqlite3_close(DB);
}
__global__ void insert_data(const char*dir){
	sqlite3* DB;
	int exit = 0;
	exit = sqlite3_open(dir,&DB);
	char* query = "INSERT INTO contacts VALUES\
	( 21279,'Yusif','IBRAHIMOV','ioseff@mail.com',0508325841),\
	( 21270,'VASIF','MAJIDOV','vasif@mail.com',0508325842),\
	( 21271,'AGHAELI','RAHIMOV','agaeli@mail.com',0508325843),\
	( 21272,'SAMIR','SADIQOV','samir@mail.com',0508325844);";
	char* message;
	exit = sqlite3_exec(DB,query,NULL,0,&message);
	if(message != NULL){
		printf("%s",message);
	}
	
	sqlite3_close(DB);

}
__global__ void select_data(const char* dir){
	sqlite3* DB;
	int exit = 0;
	exit = sqlite3_open(dir,&DB);
	char* query = " SELECT * FROM contacts;";
	char* message;
	exit = sqlite3_exec(DB,query,callback,NULL,&message);
	printf("%s",message);
}
__global__ static int  callback(void* NotUsed, int argc, char **argv, char** azColName ){
	NotUsed = 0;
    int i = threadIdx.x;
    if(i < argc){
        printf("%s : %s\n",azColName[i],argv[i]);
    }
	printf("\n");
	return 0;
}

static int  callback(void* NotUsed, int argc, char **argv, char** azColName );
int main(){

	const char*  db_dir = "new_dat.db";
    sqlite3* DB;
    hipMallocManaged(&DB)
	<<<1,SIZE>>>create_db(db_dir);
	<<<1,SIZE>>>create_table(db_dir);
	<<<1,SIZE>>>insert_data(db_dir);
    <<<1,SIZE>>>select_data(db_dir);
    
    hipFree(DB);
    hipFree(db_dir);
}
